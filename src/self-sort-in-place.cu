#include <iostream>
#include <random>
#include <cassert>
#include <hip/hip_runtime.h>
#include <ctime>
#include <cstring>

#define P (469762049 ) // 29 * 2^57 + 1ll
#define root (3)

inline long long qpow(long long x, long long y) {
    long long base = 1ll;
    while(y) {
        if (y & 1ll) base = (base * x) % P;
        x = (x * x) % P;
        y >>= 1ll;
    }
    return base;
}

inline long long inv(long long x) {
    return qpow(x, P - 2);
}

void swap(long long &a, long long &b) {
    long long tmp = a;
    a = b;
    b = tmp;
}


void NTT(long long data[], long long reverse[], long long len, long long omega) {
    // rearrange the coefficients
    for (long long i = 0; i < len; i++) {
        if (i < reverse[i]) swap(data[i], data[reverse[i]]);
    }
    
    
    for (long long stride = 1ll; stride < len; stride <<= 1ll) {
        long long gap = qpow(omega, (P - 1ll) / (stride << 1ll));
        for (long long start = 0; start < len; start += (stride << 1ll)) {
            for (long long offset = 0, w = 1ll; offset < stride; offset++, w = (gap * w) % P) {
                long long a = data[start + offset], b = w * data[start + offset + stride] % P;
                data[start + offset] = (a + b) % P;
                data[start + offset + stride] = (a - b + P) % P;
                // printf("%lld %lld\n", w, offset);
            }
        }
    }
    
}

void NTT_dif(long long data[], long long reverse[], uint log_len, long long omega) {
    uint len = 1 << log_len;
    
    for (uint i = log_len; i > 0; i--) {
        uint stride = 1 << (i - 1);
        long long gap = qpow(omega, (P - 1ll) / (stride << 1ll));
        for (uint start = 0; start < len; start += (stride << 1ll)) {
            for (uint offset = 0, w = 1ll; offset < stride; offset++, w = (gap * w) % P) {
                long long a = data[start + offset], b = data[start + offset + stride];
                data[start + offset] = (a + b) % P;
                data[start + offset + stride] = ((a - b + P) % P) * w % P;
            }
        }
        // for (int i = 0; i < len; i++) {
        //     printf("%lld ", data[i]);
        // }
        // printf("\n");
    }

    // rearrange the coefficients
    for (long long i = 0; i < len; i++) {
        if (i < reverse[i]) swap(data[i], data[reverse[i]]);
    }

}

void NTT_pro1(long long data[], uint log_len, long long omega) {
    uint len = 1 << log_len;
    
    for (uint i = log_len; i > log_len / 2; i--) {
        uint stride = 1 << (i - 1);
        long long gap = qpow(omega, (P - 1ll) / (stride << 1ll));
        for (uint start = 0; start < len; start += (stride << 1ll)) {
            for (uint offset = 0, w = 1ll; offset < stride; offset++, w = (gap * w) % P) {
                long long a = data[start + offset], b = data[start + offset + stride];
                data[start + offset] = (a + b) % P;
                data[start + offset + stride] = ((a - b + P) % P) * w % P;
            }
        }
    }

}

void NTT_pro2(long long data[], uint log_len, long long omega) {
    uint len = 1 << log_len;

    for (uint i = log_len / 2; i >= 1; i--) {
        // for (int j = 0; j < len; j++) {
        //     printf("%lld ", data[j]);
        // }
        // printf("\n");

        uint stride = 1 << (i - 1);
        uint pair_stride = 1 << (log_len - i);
        long long gap = qpow(omega, (P - 1ll) / (stride << 1));
        for (uint start = 0; start < len; start += (pair_stride << 1)) {
            for (uint offset0 = 0; offset0 < pair_stride; offset0 += (stride << 1)) {
                for (uint offset = 0, w = 1ll; offset < stride; offset++, w = (gap * w) % P) {
                    long long a = data[start + offset0 + offset];
                    long long b = data[start + offset0 + offset + stride];
                    long long c = data[start + offset0 + offset + pair_stride];
                    long long d = data[start + offset0 + offset + pair_stride + stride];
                    data[start + offset0 + offset] = (a + b) % P;
                    data[start + offset0 + offset + stride] = (c + d) % P;
                    data[start + offset0 + offset + pair_stride] = ((a - b + P) % P) * w % P;
                    data[start + offset0 + offset + pair_stride + stride] = ((c - d + P) % P) * w % P;
                }
            }
        }
    }
    // for (int i = 0; i < len; i++) {
    //         printf("%lld ", data[i]);
    //     }
    //     printf("\n");

}

__forceinline__ __device__ long long FIELD_pow_lookup(long long *omegas, uint exponent) {
    long long res = 1ll;
    uint i = 0;
    while(exponent > 0) {
        if (exponent & 1)
            res = (res * omegas[i]) % P;
        exponent = exponent >> 1;
        i++;
    }
    return res;
}

__forceinline__ __device__ long long FIELD_pow (long long base, uint exponent) {
    long long res = 1;
    while(exponent > 0) {
        if (exponent & 1)
        res = (res * base) % P;
        exponent = exponent >> 1;
        base = (base * base) % P;
    }
    return res;
}

__global__ void SSIP_NTT_stage1 (long long * x, // Source buffer
                        long long * pq, // Precalculated twiddle factors
                        long long * omegas, // [omega, omega^2, omega^4, ...]
                        uint n, // Number of elements
                        uint log_stride, // Log2 of `p` (Read more in the link above)
                        uint deg, // 1=>radix2, 2=>radix4, 3=>radix8, ...
                        uint max_deg)
{
    extern __shared__ long long u[];

    const uint lid = threadIdx.x;
    const uint index = blockIdx.x;
    const uint lgp = log_stride - deg + 1;
    const uint end_stride = 1 << lgp; //stride of the last butterfly

    // each segment is independent
    
    uint segment_start = (index >> lgp) << (lgp + deg);
    uint segment_id = index & (end_stride - 1);
    
    uint subblock_sz = 1 << (deg - 1); // # of neighbouring butterfly in the last round
    uint subblock_id = segment_id & (end_stride - 1);

    x += segment_start + subblock_id;

    uint group_id = lid & (subblock_sz - 1);

    uint gpos = group_id << (lgp + 1);
    

    u[(lid << 1)] = x[gpos];
    u[(lid << 1) + 1] = x[gpos + end_stride];

    __syncthreads();

    const uint pqshift = max_deg - deg;
    for(uint rnd = 0; rnd < deg; rnd++) {
        const uint bit = subblock_sz >> rnd;
        const uint di = lid & (bit - 1);
        const uint i0 = (lid << 1) - di;
        const uint i1 = i0 + bit;
        long long tmp = u[i0];
        u[i0] = (u[i0] + u[i1]) % P;
        u[i1] = (tmp + P - u[i1]) % P;
        if(di != 0) u[i1] = (pq[di << rnd << pqshift] * u[i1]) % P;

        __syncthreads();
    }

    // Twiddle factor
    uint k = index & (end_stride - 1);
    long long twiddle = FIELD_pow_lookup(omegas, (n >> (log_stride - deg + 1) >> deg) * k);

    long long t1 = FIELD_pow(twiddle, __brev(lid << 1) >> (32 - deg));
    long long t2 = FIELD_pow(twiddle, __brev((lid << 1) + 1) >> (32 - deg));

    // printf("%u %u\n" ,(n >> (log_stride - deg + 1) >> deg) * k * (__brev(lid << 1) >> (32 - deg)), segment_start + subblock_id+gpos);
    // printf("%u %u\n" ,(n >> (log_stride - deg + 1) >> deg) * k * (__brev((lid << 1) + 1) >> (32 - deg)), segment_start + subblock_id+gpos + end_stride);
    x[gpos] = t1 * u[(lid << 1)] % P;
    x[gpos + end_stride] = t2 * u[(lid << 1) + 1] % P;
}

__global__ void SSIP_NTT_stage2 (long long * data, // Source buffer
                        long long * pq, // Precalculated twiddle factors
                        long long * omegas, // [omega, omega^2, omega^4, ...]
                        uint log_len, // Number of elements
                        uint log_stride, // Log2 of `p` (Read more in the link above)
                        uint deg, // 1=>radix2, 2=>radix4, 3=>radix8, ...
                        uint max_deg) {
    extern __shared__ long long u[];

    uint lid = threadIdx.x;
    uint index = blockIdx.x;
    uint end_stride = 1 << (log_stride - deg + 1); //stride of the last butterfly
    uint start_pair_stride = 1 << (log_len - log_stride - 1); // the stride between the first pair of butterfly
    uint end_pair_stride = start_pair_stride << (deg - 1); // the stride between the last pair of butterfly

    // each segment is independent
    uint segment_stride = end_pair_stride << 1; // the distance between two segment
    uint segment_num = segment_stride >> (deg << 1); // # of blocks in a segment
    
    uint segment_start = index / segment_num * segment_stride;
    uint segment_id = index & (segment_num - 1);
    
    uint subblock_sz = 1 << (deg - 1); // # of neighbouring butterfly in the last round
    uint subblock_offset = (segment_id / (end_stride)) * (2 *subblock_sz * end_stride);
    uint subblock_id = segment_id & (end_stride - 1);

    data += segment_start + subblock_offset + subblock_id;

    uint group_offset = (lid / subblock_sz) * (start_pair_stride);

    uint group_id = lid & (subblock_sz - 1);

    uint gpos = group_offset + group_id * (end_stride << 1);

    u[(lid << 1)] = data[gpos];
    u[(lid << 1) + 1] = data[gpos + end_stride];
    u[(lid << 1) + (blockDim.x << 1)] = data[gpos + end_pair_stride];
    u[(lid << 1) + (blockDim.x << 1) + 1] = data[gpos + end_pair_stride + end_stride];

    __syncthreads();

    const uint pqshift = max_deg - deg;
    for(uint rnd = 0; rnd < deg; rnd++) {
       
        const uint bit = subblock_sz >> rnd;
        const uint gap = (blockDim.x << 1) >> (deg - rnd - 1);
        const uint offset = (gap) * (lid / (gap >> 1));

        const uint di = lid & (bit - 1);
        const uint i0 = (lid << 1) - di + offset;
        const uint i1 = i0 + bit;
        const uint i2 = i0 + gap;
        const uint i3 = i0 + gap + bit;
        long long a, b, c, d;
        a = u[i0], b = u[i1], c = u[i2], d = u[i3];

        u[i0] = (a + b) % P;
        u[i2] = (c + d) % P;

        u[i1] = ((a - b + P) % P);
        if(di != 0) u[i1] = (pq[di << rnd << pqshift] * u[i1]) % P;

        u[i3] = ((c - d + P) % P);
        if(di != 0) u[i3] = (pq[di << rnd << pqshift] * u[i3]) % P;

        __syncthreads();
    }

    // Twiddle factor
    uint k = index & (end_stride - 1);
    uint n = 1 << log_len;
    long long twiddle = FIELD_pow_lookup(omegas, (n >> (log_stride - deg + 1) >> deg) * k);
    // if (threadIdx.x == 0);
    // printf("%d %d\n",blockIdx.x, (n >> (log_stride - deg + 1) >> deg) * k);

    long long t1 = FIELD_pow(twiddle, lid << 1 >> deg);
    long long t2 = FIELD_pow(twiddle, ((lid << 1) + (blockDim.x <<1)) >> deg);
    // data[gpos] = u[(lid << 1)] * t1 % P;
    // data[gpos + end_pair_stride] = u[(lid << 1) + (blockDim.x << 1)] * t1 % P;
    // t1 = t1 * twiddle % P;
    // data[gpos + end_stride] = u[(lid << 1) + 1] * t1 % P;
    // data[gpos + end_pair_stride + end_stride] = u[(lid << 1) + (blockDim.x << 1) + 1] * t1 % P;

    uint a, b, c, d;
    a = __brev(lid << 1) >> (32 - (deg << 1));
    b = __brev((lid << 1) + 1) >> (32 - (deg << 1));
    c = __brev((lid << 1) + (blockDim.x << 1)) >> (32 - (deg << 1));
    d = __brev((lid << 1) + (blockDim.x << 1) + 1) >> (32 - (deg << 1));
    //printf("%u %u %u %u\n", a, b, c, d);

    data[gpos] = u[a] * t1 % P;
    data[gpos + end_stride] = u[b] * t1 % P;
    data[gpos + end_pair_stride] = u[c] * t2 % P;
    data[gpos + end_pair_stride + end_stride] = u[d] * t2 % P;
    
}

#define MAX_LOG2_RADIX 11u
#define MAX_STAGE2_RADIX 6u
void SSIP(long long *x,long long omega, uint log_n) {

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);



    uint n = 1 << log_n;

    omega = qpow(omega, (P - 1ll) / n);

    printf("%lld\n", omega);
    // for (uint i = 0; i <= n; i++) printf("%lld ", qpow(omega, i));
    // return;
    // All usages are safe as the buffers are initialized from either the host or the GPU
    // before they are read.
    // let mut src_buffer = unsafe { program.create_buffer::<F>(n)? };
    // let mut dst_buffer = unsafe { program.create_buffer::<F>(n)? };
    // The precalculated values pq` and `omegas` are valid for radix degrees up to `max_deg`
    int max_deg = std::min(MAX_LOG2_RADIX, log_n);

    // Precalculate:
    // [omega^(0/(2^(deg-1))), omega^(1/(2^(deg-1))), ..., omega^((2^(deg-1)-1)/(2^(deg-1)))]
    long long *pq, *pq_d;
    long long *omegas, *omegas_d;
    pq = new long long[1 << max_deg >> 1];
    memset (pq, 0, sizeof(long long) * (1 << max_deg >> 1));
    pq[0] = 1;
    long long twiddle = qpow(omega, ((long long)n) >> (1ll*max_deg));
    if (max_deg > 1) {
        pq[1] = twiddle;
        for (uint i = 2; i < (1 << max_deg >> 1) ; i++ ) {
            pq[i] = pq[i - 1];
            pq[i] = pq[i] *(twiddle)%P;
        }
    }
    hipMalloc(&pq_d, sizeof(long long) * (1 << max_deg >> 1));
    hipMemcpy(pq_d, pq, sizeof(long long) * (1 << max_deg >> 1), hipMemcpyHostToDevice);

    // Precalculate [omega, omega^2, omega^4, omega^8, ..., omega^(2^31)]
    omegas = new long long[32];
    memset (omegas, 0, sizeof(long long) * 32);
    omegas[0] = omega;
    for (uint i  = 1; i < 32; i++) {
        omegas[i] = omegas[i - 1] * omegas[i - 1] % P;
    }
    hipMalloc(&omegas_d, sizeof(long long) * 32);
    hipMemcpy(omegas_d, omegas, sizeof(long long) * 32, hipMemcpyHostToDevice);
    long long *res = new long long[n];

    // Specifies log2 of `p`, (http://www.bealto.com/gpu-fft_group-1.html)
    int log_p = log_n - 1;
    
    hipEventRecord(start);

    // Each iteration performs a FFT round
    while (log_p >= log_n / 2) {

        // 1=>radix2, 2=>radix4, 3=>radix8, ...
        uint deg = std::min(max_deg, (int)(log_p + 1 - log_n / 2));

        uint n = 1u << log_n;
        dim3 block(1 << (deg - 1) );
        dim3 grid(n >> deg);

        // printf("%d %d %d\n", block.x, grid.x, deg);

        SSIP_NTT_stage1 <<< grid, block, sizeof(long long) * (1 << deg) >>>(x, pq_d, omegas_d, n, log_p, deg, max_deg);

        log_p -= deg;
        // cudaMemcpy(res, x, sizeof(*res) * n, cudaMemcpyDeviceToHost);
        // for (int i = 0; i < n; i++) printf("%lld ", res[i]);
        // printf("\n");
    }
    assert (log_p == log_n / 2 - 1);
    int max_deg2 = std::min(max_deg, (int)MAX_STAGE2_RADIX);
    while (log_p >= 0) {
        // 1=>radix2, 2=>radix4, 3=>radix8, ...
        uint deg = std::min(max_deg2, log_p + 1);

        uint n = 1u << log_n;
        dim3 block1(1 << (deg << 1) >> 2);
        dim3 grid1(n / 4 / block1.x);

        // printf("%d %d %d\n", block1.x, grid1.x, deg);

        SSIP_NTT_stage2 <<< grid1, block1, sizeof(long long) * (1 << (deg << 1)) >>>(x, pq_d, omegas_d, log_n, log_p, deg, max_deg);

        log_p -= deg;
        // cudaMemcpy(res, x, sizeof(*res) * n, cudaMemcpyDeviceToHost);
        // for (int i = 0; i < n; i++) printf("%lld ", res[i]);
        // printf("\n");
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float t;
    hipEventElapsedTime(&t, start, end);
    delete [] res;

    printf("SSIP: %fms\n", t);
    free(pq);
    free(omegas);
    hipFree(pq_d);
    hipFree(omegas_d);
}

int main() {
    long long *data, *reverse, *data_copy;
    long long l,length = 1ll;
    int bits = 0;

    //scanf("%lld", &l);
    l = qpow(2, 24);

    while (length < l) {
        length <<= 1ll;
        bits ++;
    }

    assert(length == (1ll << bits));
    assert(bits <= 57);

    data = new long long[length];
    data_copy = new long long[length];
    reverse = new long long [length];

    for (long long i = 0; i < length; i++) {
        reverse[i] = (reverse[i >> 1ll] >> 1ll) | ((i & 1ll) << (bits - 1ll) ); //reverse the bits
    }

    std::random_device rd;
    std::mt19937_64 gen(rd());
    for (long long i = 0; i < length; i++) {
        data[i] = i; std::abs((long long)gen()) % P;
        data_copy[i] = data[i];
    }

    // cpu implementation
    {
        clock_t start = clock();

        NTT(data, reverse, length, root);

        clock_t end = clock();
        printf("cpu: %lfms\n",(double)(end - start) / CLOCKS_PER_SEC * 1000);
    }

    long long *data_new;
    data_new = new long long[length];
    memcpy(data_new, data_copy, length * sizeof(long long));

    // Cpu SSIP
    {
        clock_t start = clock();

        NTT_pro1(data_new, bits, root);
        NTT_pro2(data_new, bits, root);
        // NTT_dif(data_new, reverse, bits, root);

        clock_t end = clock();
        printf("cpu: %lfms\n",(double)(end - start) / CLOCKS_PER_SEC * 1000);

        for (long long i = 0; i < length; i++) {
            if (data_new[i] != data[i]) {
                printf("error\n");
                return 0;
            }
        }
        printf("correct\n");
    }

    long long * data_d;
    hipMalloc(&data_d, sizeof(long long) * length);
    hipMemcpy(data_d, data_copy, sizeof(long long) * length, hipMemcpyHostToDevice);

    SSIP(data_d, root, bits);

    hipMemcpy(data_new, data_d, length * sizeof(long long), hipMemcpyDeviceToHost);

    for (long long i = 0; i < length; i++) {
        if (data_new[i] != data[i]) {
            printf("error\n");
            return 0;
        }
    }
    printf("correct\n");
    // run_kernel(data_d, bits);
    return 0;
}